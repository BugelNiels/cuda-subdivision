#include "hip/hip_runtime.h"
#include "kernelInvoker.cuh"
#include "kernels/meshRefinement.cuh"
#include "kernels/quadRefinement.cuh"
#include "kernels/optimisedQuadRef.cuh"
#include "util/util.cuh"
#include "util/deviceCommunication.cuh"

#include "stdio.h"
#include "math.h"

#define USE_OPTIMIZED_KERNEL 1

// swaps pointers
void meshSwap(DeviceMesh **prevMeshPtr, DeviceMesh **newMeshPtr) {
  DeviceMesh *temp = *prevMeshPtr;
  *prevMeshPtr = *newMeshPtr;
  *newMeshPtr = temp;
}

DeviceMesh performSubdivision(DeviceMesh* input, DeviceMesh* output, int subdivisionLevel, Mesh* mesh) {
  hipError_t cuda_ret;
  hipEvent_t start, stop;

  int h0 = mesh->numHalfEdges;
  int v0 = mesh->numVerts;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  
	DeviceMesh* in = toDevicePointer(input);
	DeviceMesh* out = toDevicePointer(output);

  dim3 dim_grid, dim_block;

  // each thread takes 1 half edge
  dim_block.x = BLOCK_SIZE;
  dim_block.y = dim_block.z = 1;
  dim_grid.y = dim_grid.z = 1;

  printf("\n------------------\nPerforming subdivision\n...\n");

  // device must be synced before this point
  hipEventRecord(start);
  // all the stuff before this can be pre-allocated/pre-calculated

  copyHostToDeviceMesh(mesh, input);	

  int startLevel = 0;
  int v = v0;
  if(!mesh->isQuad) {
    dim_grid.x = MIN((h0 - 1) / BLOCK_SIZE + 1, MAX_GRID_SIZE);
    resetMesh<<<dim_grid, dim_block>>>(in, out);
    refineTopology<<<dim_grid, dim_block>>>(in, out);
    facePoints<<<dim_grid, dim_block>>>(in, out);
    edgePoints<<<dim_grid, dim_block>>>(in, out);
    vertexPoints<<<dim_grid, dim_block>>>(in, out);
    meshSwap(&in, &out);
    startLevel = 1;
    v = mesh->numVerts + mesh->numFaces + mesh->numEdges;
  } 

  for (int d = startLevel; d < subdivisionLevel; d++) {
    // each thread covers 1 half edge. Number of half edges can be much greater than blockdim * gridDim. 
    int he = pow(4, d) * h0;
    dim_grid.x = MIN((he - 1) / BLOCK_SIZE + 1, MAX_GRID_SIZE);
    if(USE_OPTIMIZED_KERNEL) {
      
      resetMesh<<<dim_grid, dim_block>>>(in, out);
      optimisedSubdivide<<<dim_grid, dim_block>>>(in, out, v);
    } else {
      resetMesh<<<dim_grid, dim_block>>>(in, out);
      quadRefineTopology<<<dim_grid, dim_block>>>(in, out);
      quadFacePoints<<<dim_grid, dim_block>>>(in, out);
      quadEdgePoints<<<dim_grid, dim_block>>>(in, out);
      quadVertexPoints<<<dim_grid, dim_block>>>(in, out, v);
    }
    // result is in "out"; after this swap, the result is in "in"
    meshSwap(&in, &out);
  }
  hipEventRecord(stop);
  cuda_ret = hipDeviceSynchronize();
  cudaErrCheck(cuda_ret, "Unable to execute kernel");
  
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Execution took: %lf msec\n------------------\n\n", milliseconds);
  DeviceMesh m =  devicePointerToHostMesh(in);
  
  hipFree(in);
  hipFree(out);
  return m;
  
}