#include "hip/hip_runtime.h"
#include "quadRefinement.cuh"
#include "../util/util.cuh"
#include "kernelUtil/kernelUtils.cuh"
#include "stdio.h"

__global__ void optimisedSubdivide(DeviceMesh* in, DeviceMesh* out, int v0) {
    
    __shared__ float facePointsX[FACES_PER_BLOCK];
    __shared__ float facePointsY[FACES_PER_BLOCK];
    __shared__ float facePointsZ[FACES_PER_BLOCK];

    int vd = in->numVerts;
    int fd = in->numFaces;
    int ed = in->numEdges;

    int ti = threadIdx.x / 4;
    int t2 = threadIdx.x % 4;

    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int h = start; h < in->numHalfEdges; h += stride) {
        // not all threads in the warp execute this, but it should eliminate the need for thread sync
        if(t2 == 0) {
            // reset shared memory
            facePointsX[ti] = 0;
            facePointsY[ti] = 0;
            facePointsZ[ti] = 0;
        }
        // edge refinement
        int hp = prev(h);
        int he = in->edges[h];
        int v = in->verts[h];
        int ht = in->twins[h];

        out->twins[4 * h] = ht < 0 ? -1 : 4 * next(ht) + 3;
        out->twins[4 * h + 1] = 4 * next(h) + 2;
        out->twins[4 * h + 2] = 4 * hp + 1;
        out->twins[4 * h + 3] = 4 * in->twins[hp];

        out->verts[4 * h] = v;
        out->verts[4 * h + 1] = vd + fd + he;
        out->verts[4 * h + 2] = vd + face(h);
        out->verts[4 * h + 3] = vd + fd + in->edges[hp];

        out->edges[4 * h] = h > ht ? 2 * he : 2 * he + 1;
        out->edges[4 * h + 1] = 2 * ed + h;
        out->edges[4 * h + 2] = 2 * ed + hp;
        out->edges[4 * h + 3] = hp > in->twins[hp] ? 2 * in->edges[hp] + 1 : 2 * in->edges[hp];
        
        // face points
        float invX = in->xCoords[v];
        float invY = in->yCoords[v];
        float invZ = in->zCoords[v];

        atomicAdd(&facePointsX[ti], invX / 4.0f);
        atomicAdd(&facePointsY[ti], invY / 4.0f);
        atomicAdd(&facePointsZ[ti], invZ / 4.0f);

        // edge points
        float x, y, z;

        int vNext = in->verts[next(h)];
        float edgex = (invX + in->xCoords[vNext]) / 2.0f;
        float edgey = (invY + in->yCoords[vNext]) / 2.0f;
        float edgez = (invZ + in->zCoords[vNext]) / 2.0f;
        
        // boundary edge point
        if(ht < 0) {
            x = edgex;
            y = edgey;
            z = edgez;      
        } else {
            // average the vertex of this vertex and the face point
            x = (invX + facePointsX[ti]) / 4.0f;
            y = (invY + facePointsY[ti]) / 4.0f;
            z = (invZ + facePointsZ[ti]) / 4.0f;
        }    
        int j = vd + fd + he;
        atomicAdd(&out->xCoords[j], x);
        atomicAdd(&out->yCoords[j], y);
        atomicAdd(&out->zCoords[j], z);

        float n = valenceQuad(h, in);
        if(n > 0) {
            float n2 = n * n;
            x = (2 * edgex + facePointsX[ti] + (n - 3) * invX) / n2;
            y = (2 * edgey + facePointsY[ti] + (n - 3) * invY) / n2;
            z = (2 * edgez + facePointsZ[ti] + (n - 3) * invZ) / n2;
            atomicAdd(&out->xCoords[v], x);
            atomicAdd(&out->yCoords[v], y);
            atomicAdd(&out->zCoords[v], z);
        } else if(ht < 0) {
            // boundary vertex point
            x = (edgex + invX) / 4.0f;
            y = (edgey + invY) / 4.0f;
            z = (edgez + invZ) / 4.0f;
            atomicAdd(&out->xCoords[v], x);
            atomicAdd(&out->yCoords[v], y);
            atomicAdd(&out->zCoords[v], z);

            // do similar thing for the next vertex
            x = (edgex + in->xCoords[vNext]) / 4.0f;
            y = (edgey + in->yCoords[vNext]) / 4.0f;
            z = (edgez + in->zCoords[vNext]) / 4.0f;
            atomicAdd(&out->xCoords[vNext], x);
            atomicAdd(&out->yCoords[vNext], y);
            atomicAdd(&out->zCoords[vNext], z);
        }
        if(t2 == 0) {
            int ind = vd + face(h);
            out->xCoords[ind] = facePointsX[ti];
            out->yCoords[ind] = facePointsY[ti];
            out->zCoords[ind] = facePointsZ[ti];
        }
    }  
}
